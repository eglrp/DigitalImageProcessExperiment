#include "hip/hip_runtime.h"
#ifndef  __MEDIANFILTER_CU_
#define  __MEDIANFILTER_CU_

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#define datasize 100

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__device__ int rs_normaliseXY(int x, int y, int max_x, int max_y) {
	if (x >= max_x || x <= 0 || y >= max_y || y <= 0) //�����߽�Ĳ����ú�ɫ���
		return 0;
	return x;
}
 __device__ int rs_checkColorSpace(double x) {
	if (x > 255)
		return 255;
	if (x < 0)
		return 0;
	return x;
}
__device__ float rs_BiCubicPoly(float x)
{
	float abs_x = abs(x);
	float a = -0.5;
	if (abs_x <= 1.0)
	{
		return (a + 2)*pow(abs_x, 3) - (a + 3)*pow(abs_x, 2) + 1;
	}
	else if (abs_x < 2.0)
	{
		return a*pow(abs_x, 3) - 5 * a*pow(abs_x, 2) + 8 * a*abs_x - 4 * a;
	}
	else
		return 0.0;
}
__global__ void Rotate_And_Scale(int *In, int *Out, int Width, int Height,double angle,double factor)
{
	double M_PI = 3.1415926;
	//  Degrees-to-radians constant 
	const double  DegreesToRadians = M_PI / 180.0;
	//�������
	double theta = angle * DegreesToRadians;// ��ת�ĽǶ�
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x <= Width && x >= 0 && y <= Height && y >= 0)
	{

		//������ת���v,w���п��ܴ���С��
		double real_v = x*cos(theta) + y*sin(theta) + cos(theta)*(-1)*(Width / 2) + sin(theta)*(-1)*(Height / 2) ;
		double real_w = (-1)*x*sin(theta) + y*cos(theta) + cos(theta)*(-1)*(Height / 2) - sin(theta)*(-1)*(Width / 2) ;
		//�����ź�ƽ��
		real_v = real_v / factor - (-1)*Width / 2;
		real_w = real_w / factor - (-1)*Height / 2;
		//��ÿһ��λ�ã�x,y��ʹ��(v,w) = T^(-1)(x,y)��������ͼ���е���Ӧλ�� 
		double value = 0;
		int v, w;
		//ȡ��������
		v = floor(real_v);
		w = floor(real_w);
		if ((value = rs_normaliseXY(v, w, Width, Height)) != 0)//��û�����߽�Ļ�
			if (v >= 2 && v < Width - 2 && w >= 2 && w < Height - 2) {//��ֹԽ��														
				//ʵ��˫�����ڲ�
				value = 0;
				//4*4��������
				for (int i = -1; i < 3; i++)
					for (int j = -1; j < 3; j++)
						value += (*(In + Width * (w + i) + (v + j)))*rs_BiCubicPoly(real_v - v - j)*rs_BiCubicPoly(real_w - w - i);
			}
		value = rs_checkColorSpace(value);
		Out[y* Width + x] = value;
	}
}

extern "C" void rotate_and_scale_host(int *pixel, int Width, int Height,double angle, double factor)
{
	int *pixelIn, *pixelOut;
	dim3 dimBlock(32, 32);
	dim3 dimGrid((Width + dimBlock.x - 1) / dimBlock.x, (Height + dimBlock.y -
		1) / dimBlock.y);
	checkCudaErrors(hipMalloc((void**)&pixelIn, sizeof(int) * Width * Height));
	checkCudaErrors(hipMalloc((void**)&pixelOut, sizeof(int) * Width * Height));

	checkCudaErrors(hipMemcpy(pixelIn, pixel, sizeof(int) * Width * Height, hipMemcpyHostToDevice));

	Rotate_And_Scale << <dimGrid, dimBlock >> > (pixelIn, pixelOut, Width , Height, angle,factor);

	checkCudaErrors(hipMemcpy(pixel, pixelOut, sizeof(int) * Width * Height, hipMemcpyDeviceToHost));


	hipFree(pixelIn);
	hipFree(pixelOut);
}

#endif // ! __MEDIANFILTER_KERNEL_CU_