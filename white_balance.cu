#include "hip/hip_runtime.h"
#ifndef  __MEDIANFILTER_CU_
#define  __MEDIANFILTER_CU_

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#define datasize 100

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}
__device__ int wb_checkColorSpace(double x) {
	if (x > 255)
		return 255;
	if (x < 0)
		return 0;
	return x;
}

__global__ void white_balance(int *In, int *Out, int Width, int Height, double color_sum, double RGB_sum)
{
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//��Ҫ������RGB����������
	double K = (RGB_sum) / (3 * color_sum);
	if (x <= Width && x >= 0 && y <= Height && y >= 0)
	{
		Out[y* Width + x] = wb_checkColorSpace((*(In + Width * y + x))*K);
	}
}

extern "C" void white_balance_host(int *pixel, int Width, int Height, double color_sum, double RGB_sum)
{
	int *pixelIn, *pixelOut;
	dim3 dimBlock(32, 32);
	dim3 dimGrid((Width + dimBlock.x - 1) / dimBlock.x, (Height + dimBlock.y -
		1) / dimBlock.y);
	checkCudaErrors(hipMalloc((void**)&pixelIn, sizeof(int) * Width * Height));
	checkCudaErrors(hipMalloc((void**)&pixelOut, sizeof(int) * Width * Height));

	checkCudaErrors(hipMemcpy(pixelIn, pixel, sizeof(int) * Width * Height, hipMemcpyHostToDevice));

	white_balance << <dimGrid, dimBlock >> > (pixelIn, pixelOut, Width , Height, color_sum, RGB_sum);

	checkCudaErrors(hipMemcpy(pixel, pixelOut, sizeof(int) * Width * Height, hipMemcpyDeviceToHost));


	hipFree(pixelIn);
	hipFree(pixelOut);
}

#endif // ! __MEDIANFILTER_KERNEL_CU_